#include "hip/hip_runtime.h"
#include "io.h"
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
#include <iostream>

void check_cuda_error(const char* msg);

// Lennard-Jones force calculation kernel
__global__ void compute_forces_kernel(Particle* particles, int N, float sigma, float epsilon) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    float3 force = make_float3(0, 0, 0);
    float3 pi = particles[i].pos;

    for (int j = 0; j < N; ++j) {
        if (i == j) continue;
        float3 pj = particles[j].pos;
        float3 rij = {pj.x - pi.x, pj.y - pi.y, pj.z - pi.z};
        float r2 = rij.x*rij.x + rij.y*rij.y + rij.z*rij.z + 1e-8f;
        float r6 = r2 * r2 * r2;
        float r12 = r6 * r6;
        float sig6 = sigma*sigma*sigma*sigma*sigma*sigma;
        float sig12 = sig6 * sig6;
        float fmag = 24 * epsilon * (2 * sig12 / r12 - sig6 / r6) / r2;
        force.x += fmag * rij.x;
        force.y += fmag * rij.y;
        force.z += fmag * rij.z;
    }
    particles[i].acc.x = force.x / particles[i].mass;
    particles[i].acc.y = force.y / particles[i].mass;
    particles[i].acc.z = force.z / particles[i].mass;
}

void launch_compute_forces(Particle* d_particles, int N, float sigma, float epsilon) {
    int block = 128;
    int grid = (N + block - 1) / block;
    compute_forces_kernel<<<grid, block>>>(d_particles, N, sigma, epsilon);
    check_cuda_error("compute_forces_kernel");
}

// Velocity Verlet integration kernel
__global__ void integrate_first_half_kernel(Particle* particles, int N, float dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    // Velocity half-step update
    particles[i].vel.x += 0.5f * particles[i].acc.x * dt;
    particles[i].vel.y += 0.5f * particles[i].acc.y * dt;
    particles[i].vel.z += 0.5f * particles[i].acc.z * dt;

    // Position full-step update
    particles[i].pos.x += particles[i].vel.x * dt;
    particles[i].pos.y += particles[i].vel.y * dt;
    particles[i].pos.z += particles[i].vel.z * dt;
}

void launch_integrate_first_half(Particle* d_particles, int N, float dt) {
    int block = 128;
    int grid = (N + block - 1) / block;
    integrate_first_half_kernel<<<grid, block>>>(d_particles, N, dt);
    check_cuda_error("integrate_first_half_kernel");
}

__global__ void integrate_second_half_kernel(Particle* particles, int N, float dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    // Final velocity half-step update using new acceleration
    particles[i].vel.x += 0.5f * particles[i].acc.x * dt;
    particles[i].vel.y += 0.5f * particles[i].acc.y * dt;
    particles[i].vel.z += 0.5f * particles[i].acc.z * dt;
}

void launch_integrate_second_half(Particle* d_particles, int N, float dt) {
    int block = 128;
    int grid = (N + block - 1) / block;
    integrate_second_half_kernel<<<grid, block>>>(d_particles, N, dt);
    check_cuda_error("integrate_second_half_kernel");
}


void print_particles(const std::vector<Particle>& particles, int max_print) {
    for (int i = 0; i < std::min((int)particles.size(), max_print); ++i) {
        const auto& p = particles[i];
        std::cout << "Particle " << i << ": pos=(" << p.pos.x << "," << p.pos.y << "," << p.pos.z
                  << ") vel=(" << p.vel.x << "," << p.vel.y << "," << p.vel.z
                  << ") mass=" << p.mass << std::endl;
    }
}

void check_cuda_error(const char* msg) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error after " << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}